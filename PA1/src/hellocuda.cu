/*
  In this program we will be using the GPU to add two square matrices. 

  This is the main program. You should also look at the header matgpuadd.h
  matcpuadd.h, and matdefine.h for the important declarations, and then look
  at matgpuadd.cu, matcpuadd.cu, and matdefine.cu to see how the methods were
  defined.

  AUTHOR: Steven Fisher
  CLASS: CS 791-GPU Computing
  ASSIGNMENT: PA1
 */

#include <iostream>

#include "matgpuadd.h"
#include "matcpuadd.h"
#include "matdefine.h"


int main() {
/*
This section is for the declaration of the variables that will be used
in our implementation of the matrix addition.
*/

	char check;

	int Grid_Dim_x=1, Grid_Dim_y=1;		//Grid structure values
	int Block_Dim_x=1, Block_Dim_y=1;	//Block structure values

	int numThreads_x;  			// number of threads available in device, each dimension
	int numThreads_block;			// number of threads in a block

	int N = 10;  				// size of array in each dimension
	int *a,*b,*c,*d, *e;
	int *dev_a, *dev_b, *dev_c, *dev_d;
	int size;					// number of elements in the matrices

	hipEvent_t start, stop, start_stride, stop_stride, throughstart, throughstop, start_mem_cpy, stop_mem_cpy;     		// using cuda events to measure time
	float elapsed_time_gpu, elapsed_time_cpu, through_total, elapsed_mem, elapsed_stride;

/*
This section specifies the size limitations and allows the user to
specify the size of the matrices, the number of blocks used and the
number of threads per block to use.
*/

	std::cout << "Maximum number of threads per block = 1024" << std::endl;
	std::cout << "Maximum sizes of the x and y dimensions of the thread block = 1024" << std::endl;
	std::cout << "Maximum size of each dimension of grid of thread blocks = 65535" << std::endl;
do {	
	do {
	   std::cout << "Enter the value(size of matrix) for N (N <= 20000): ";
	   std::cin >> N;

	   if (N < 1) {
	      std::cout << "Error -- N has to be greater than 0!" << std::endl;
	   }
	   else if (N > 20000) {
 	      std::cout << "Error -- N has to be less than or equal to 1000!" << std::endl;
	   }
	} while ( N < 10 || N > 20000);
	
	do {//Using a do while loop, since we want it to run at least once.
		std::cout << "Enter number of blocks per grid that will be used in both the x and y dimensions: ";
		std::cin >> Grid_Dim_x;

		Grid_Dim_y = Grid_Dim_x;  // square grid

		std::cout << "Enter number of threads that will used per block in both the x and y dimensions, currently " << Block_Dim_x << " (Needs to be < 32): ";
		std::cin >> Block_Dim_x;

		Block_Dim_y = Block_Dim_x;	//square blocks

		numThreads_x = Grid_Dim_x * Block_Dim_x;		// total number of threads in x dimension
		
		numThreads_block = Block_Dim_x * Block_Dim_y;	// number of threads in a block

		if (numThreads_x < N) {
		   std::cout <<"Error -- number of threads in the x or y dimensions is less than thenumber of elements in matrix!" << std::endl;
		}
		else if (numThreads_block > 1024) {
		     std::cout << "Error -- there are too many threads in block!" << std::endl;
		}

	} while (numThreads_x < N || numThreads_block > 1024);

	dim3 Grid(Grid_Dim_x, Grid_Dim_y);	//Grid structure
	dim3 Block(Block_Dim_x,Block_Dim_y);	//Number of threads per block.

/*
This section will dynamically allocate the memory needed
for the matrices in both the cpu and gpu calculations.
Here we will also be using fillMatrices from matdefine in
order to populate our two matrices.
*/

	size = N * N * sizeof(int);		// number of bytes in total in arrays, this is needed in both malloc and hipMalloc

	a = (int*) malloc(size);		// Dynamically allocates the memory for the matrices on the host
	b = (int*) malloc(size);
	c = (int*) malloc(size);		// this will hold the results from the GPU calculation
	d = (int*) malloc(size);		// this will hold the results from from the CPU calculation
	e = (int*) malloc(size);

	hipMalloc((void**)&dev_a, size);	// allocate the memory for the matrices on the device
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);
	hipMalloc((void**)&dev_d, size);

	fillMatrices(a,b,N);			// used to generate the arrays, found in matdefine.cu
	
	std::cout << "Array A" << std::endl;
	printMatrix(a, N);			// used to display matrix A, used in order to verify what was in the matrix for debugging
	std::cout << "Array B" << std::endl;
	printMatrix(b, N);			// used to display matrix B, used in order to verify what was in the matrix for debugging

/*
In this section we will be performing the nececcary steps in
order to run our computaion on the GPU. The hipEventCreate is
used to created the events for our timers. hipMemcpy is used
to copy the matrix from the host to the device, which is then
used in the matgpuadd function, which used the entered results
from before, to specify the number of blocks and the number of
threads per block that will be used on the GPU
*/

  	hipEventCreate(&start);     		// Creates the event for the start timer
	hipEventCreate(&stop);			// Creates the event for the stop timer
	hipEventCreate(&throughstart);		// Creates the event for the start timer for the throughput
	hipEventCreate(&throughstop);		// Creates the event for the stop timer for the throughput
  	hipEventCreate(&start_stride);     		// Creates the event for the start timer
	hipEventCreate(&stop_stride);			// Creates the event for the stop timer
	hipEventCreate(&start_mem_cpy);     		// Creates the event for the start timer
	hipEventCreate(&stop_mem_cpy);			// Creates the event for the stop timer

	hipEventRecord(start_mem_cpy, 0);
 	hipMemcpy(dev_a, a , size ,hipMemcpyHostToDevice); //copies the information for matrix a to dev_a on the device
	hipMemcpy(dev_b, b , size ,hipMemcpyHostToDevice);
	hipEventRecord(stop_mem_cpy, 0);
	hipEventSynchronize(stop_mem_cpy);
	hipEventElapsedTime(&elapsed_mem, start_mem_cpy, stop_mem_cpy);

	hipEventRecord(start, 0);
	hipEventRecord(throughstart, 0); //records the start time for the throughput
	matgpuadd<<<Grid,Block>>>(dev_a,dev_b,dev_c,N);
	hipDeviceSynchronize();
	hipEventRecord(throughstop, 0);
	hipEventSynchronize(throughstop); //records the stop time for the throughput
	hipEventElapsedTime(&through_total, throughstart, throughstop);
	

	hipMemcpy(c,dev_c, size ,hipMemcpyDeviceToHost); //copies the results from the addition from the device to the host

	hipEventRecord(stop, 0);     	// records the stop time
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_gpu, start, stop ); //stores the elapsed time for the gpu
	


	hipEventRecord(start_stride, 0);
	matgpuadd_stride<<<Grid,Block>>>(dev_a,dev_b,dev_d,N);
	hipDeviceSynchronize();
	hipEventRecord(stop_stride, 0);
	hipEventSynchronize(stop_stride);

	hipMemcpy(e,dev_d, size ,hipMemcpyDeviceToHost); //copies the results from the addition from the device to the host
	hipEventElapsedTime(&elapsed_stride, start_stride, stop_stride);

	std::cout << "Time needed to calculate the results on the GPU: " << elapsed_time_gpu + elapsed_mem << " ms." << std::endl;  // print out elapsed time for gpu
	std::cout << "Throughput for gpu: " << N * N * through_total * 1000 << " calculations per second" << std::endl; // print out throughput for gpu.
	std::cout << "Time needed to calculate with striding: " << elapsed_stride + elapsed_mem << " ms." << std::endl;
	

/*
In this section we will be perofming the necessary steps
to run the sequential computations on the CPU
*/

	hipEventRecord(start, 0);	// records the start time

	matcpuadd(a,b,d,N);		// do calculation on the cpu

	hipEventRecord(stop, 0);     	// records the end time end time for cpu calculation
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_cpu, start, stop ); //store the elaspsed time for the cpu

	printMatrix(c,N);
	printMatrix(d,N);
	printMatrix(e,N);

	std::cout << "Time needed to calculate the results on the CPU: " << elapsed_time_cpu << " ms." << std::endl;  // print out elapsed time for the cpu

        std::cout << std::endl; 
	std::cout << "Checking if the results from the cpu calculation = gpu  calculation" << std::endl;

	for(int i = 0;i < N*N;i++) {  // checking if the matrix from the gpu is the same as cpu
		if (c[i] != d[i] ) { 
			std::cout << "ERROR results are not equal" << std::endl;
			break;
		}
	}
	
	//prints out the speedup for the gpu as compared to cpu.
	std::cout << "Speedup on GPU as compared to CPU without Stride= " << ((float) elapsed_time_cpu / ((float) elapsed_time_gpu + (float) elapsed_mem)) << std::endl;
	std::cout << "Speedup on GPU as compared to CPU with Stride= " << ((float) elapsed_time_cpu / ((float) elapsed_stride + (float) elapsed_mem)) << std::endl;


/*
Performing methods to free allocated memory
*/
	free(a);
	free(b);
	free(c);
	free(d);
	free(e);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_d);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipEventDestroy(throughstart);
	hipEventDestroy(throughstop);
	hipEventDestroy(start_stride);
	hipEventDestroy(stop_stride);
	hipEventDestroy(start_mem_cpy);
	hipEventDestroy(stop_mem_cpy);	
	std::cout << "To continue type c, to end press ctrl-z" << std::endl;
	std::cin >> check;
} while(check == 'c');
	return 0;
}
