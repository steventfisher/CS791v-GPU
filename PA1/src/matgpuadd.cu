#include "hip/hip_runtime.h"

#include "matgpuadd.h"

__global__ void matgpuadd(int *a, int *b, int *c, int N) {

  int columns = blockIdx.x * blockDim.x + threadIdx.x;
  int rows = blockIdx.y * blockDim.y + threadIdx.y;
  
  if(columns < N && rows < N) {
	c[rows * N + columns] =  a[rows * N + columns] + b[rows * N + columns];
  }
}

__global__ void matgpuadd_stride(int *a, int *b, int *c, int N) {

//  int columns = blockIdx.x * blockDim.x + threadIdx.x;
//  int rows = blockIdx.y * blockDim.y + threadIdx.y;
  
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < N;
       i += blockDim.x * gridDim.x
       ){
	c[i] =  a[i] + b[i];
  }
}