#include "hip/hip_runtime.h"

#include "matgpuadd.h"

__global__ void matgpuadd(int *a, int *b, int *c, int N) {

  int columns = blockIdx.x * blockDim.x + threadIdx.x;
  int rows = blockIdx.y * blockDim.y + threadIdx.y;

  if(columns < N && rows < N) {
	c[rows * N + columns] =  a[rows * N + columns] + b[rows * N + columns];
  }
}
