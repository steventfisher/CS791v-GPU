#include "hip/hip_runtime.h"

#include "knngp.h"
#include <math.h>

    __global__ void knnGpu(float *A, float *B, int Size)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    __syncthreads();
    
    float tmpdist = 0;
    for(int i = 1; i < Size*Size; i += 1){
       B[i] = 2;
    }
    
  
}