#include "hip/hip_runtime.h"

#include "matgpuadd.h"

__global__ void matgpuadd(int *a, int *b, int *c, int N) {

//  int columns = blockIdx.x * blockDim.x + threadIdx.x;
//  int rows = blockIdx.y * blockDim.y + threadIdx.y;
  
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < N;
       i += blockDim.x * gridDim.x
       ){
       for(int j = blockIdx.y*blockDim.y + threadIdx.y;
       j < N;
       j += blockDim.y * gridDim.y
       ){
	c[i*N + j] =  a[i*N + j] + b[i*N + j];
      }
  }
}