#include "hip/hip_runtime.h"

#include "matgpumult.h"

__global__ void matgpumult(int *a, int *b, int *c, int N) {
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    __syncthreads();

    //int sum = 0;
    if(col < N && row < N)
    {
	int sum = 0;
        for(int k = 0; k < N; ++k) {
	    sum +=  (a[row * N + k] * b[k * N + col]);
        }
        c[row * N + col] = sum;
	__syncthreads();

    }
}