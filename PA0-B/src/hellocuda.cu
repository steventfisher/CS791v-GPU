/*
  This program demonstrates the basics of working with cuda. We use
  the GPU to add two arrays. We also introduce cuda's approach to
  error handling and timing using cuda Events.

  This is the main program. You should also look at the header add.h
  for the important declarations, and then look at add.cu to see how
  to define functions that execute on the GPU.
 */

#include <iostream>

#include "add.h"

int main() {
  
  // Arrays on the host (CPU)
  int *a, *b, *c;
  
  hipMallocManaged( (void**) &a, N * sizeof(int));
  hipMallocManaged( (void**) &b, N * sizeof(int));
  hipMallocManaged( (void**) &c, N * sizeof(int));

  // These lines just fill the host arrays with some data so we can do
  // something interesting. Well, so we can add two arrays.
  for (int i = 0; i < N; ++i) {
    a[i] = i;
    b[i] = i;
  }

  /*
    FINALLY we get to run some code on the GPU. At this point, if you
    haven't looked at add.cu (in this folder), you should. The
    comments in that file explain what the add function does, so here
    let's focus on how add is being called. The first thing to notice
    is the <<<...>>>, which you should recognize as _not_ being
    standard C. This syntactic extension tells nvidia's cuda compiler
    how to parallelize the execution of the function. We'll get into
    details as the course progresses, but for we'll say that <<<N,
    1>>> is creating N _blocks_ of 1 _thread_ each. Each of these
    threads is executing add with a different data element (details of
    the indexing are in add.cu). 

    In larger programs, you will typically have many more blocks, and
    each block will have many threads. Each thread will handle a
    different piece of data, and many threads can execute at the same
    time. This is how cuda can get such large speedups.
   */
  add<<<N, 1>>>(a, b, c);
  hipDeviceSynchronize();

  /*
    Let's check that the results are what we expect.
   */
  for (int i = 0; i < N; ++i) {
    if (c[i] != a[i] + b[i]) {
      std::cerr << "Oh no! Something went wrong. You should check your cuda install and your GPU. :(" << std::endl;

      // clean up device pointers - just like free in C. We don't have
      // to check error codes for this one.
      hipFree(a);
      hipFree(b);
      hipFree(c);
      exit(1);
    }
  }

  /*
    Let's let the user know that everything is ok and then display
    some information about the times we recorded above.
   */
  std::cout << "Yay! Your program's results are correct." << std::endl;
  
  // Cleanup in the event of success.
  hipFree(a);
  hipFree(b);
  hipFree(c);

}
