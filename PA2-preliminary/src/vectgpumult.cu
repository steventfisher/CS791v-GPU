#include "hip/hip_runtime.h"

#include "vectgpumult.h"

__global__ void vectgpumult(int *a, int *b, int *c, int N) {

  __shared__ float cache[threadsPerBlock];
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int cacheIndex = threadIdx.x;
  float temp = 0;
  
  while (tid < N) {
      temp += a[tid] * b[tid]
      tid += stride;
  }

  cache[cacheIndex] = temp;
}