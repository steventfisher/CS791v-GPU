/*
  In this program we will be using the GPU to add two square matrices. 

  This is the main program. You should also look at the header matgpuadd.h
  matcpuadd.h, and matdefine.h for the important declarations, and then look
  at matgpuadd.cu, matcpuadd.cu, and matdefine.cu to see how the methods were
  defined.

  AUTHOR: Steven Fisher
  CLASS: CS 791-GPU Computing
  ASSIGNMENT: PA2
 */

#include <iostream>

#include "matgpumult.h"
#include "matcpumult.h"
#include "matdefine.h"


int main() {
/*
This section is for the declaration of the variables that will be used
in our implementation of the matrix addition.
*/

	char check;

	int Grid_Dim_x=1, Grid_Dim_y=1;		//Grid structure values
	int Block_Dim_x=1, Block_Dim_y=1;	//Block structure values

	int numThreads_x;  			// number of threads available in device, each dimension
	int numThreads_block;			// number of threads in a block

	int N = 10;  				// size of array in each dimension
	int *a,*b,*c,*d;
/*
This section specifies the size limitations and allows the user to
specify the size of the matrices, the number of blocks used and the
number of threads per block to use.
*/

	std::cout << "Maximum number of threads per block = 1024" << std::endl;
	std::cout << "Maximum sizes of the x and y dimensions of the thread block = 1024" << std::endl;
	std::cout << "Maximum size of each dimension of grid of thread blocks = 65535" << std::endl;
do {	
	do {
	   std::cout << "Enter the value(size of matrix) for N (N <= 20000): ";
	   std::cin >> N;

	   if (N < 1) {
	      std::cout << "Error -- N has to be greater than 0!" << std::endl;
	   }
	   else if (N > 20000) {
 	      std::cout << "Error -- N has to be less than or equal to 1000!" << std::endl;
	   }
	} while ( N < 10 || N > 20000);
	
	do {//Using a do while loop, since we want it to run at least once.
		std::cout << "Enter number of blocks per grid that will be used in both the x and y dimensions: ";
		std::cin >> Grid_Dim_x;

		Grid_Dim_y = Grid_Dim_x;  // square grid

		std::cout << "Enter number of threads that will used per block in both the x and y dimensions, currently " << Block_Dim_x << " (Needs to be < 32): ";
		std::cin >> Block_Dim_x;

		Block_Dim_y = Block_Dim_x;	//square blocks

		numThreads_x = Grid_Dim_x * Block_Dim_x;		// total number of threads in x dimension
		
		numThreads_block = Block_Dim_x * Block_Dim_y;	// number of threads in a block

		if (numThreads_x < N) {
		   std::cout <<"Error -- number of threads in the x or y dimensions is less than thenumber of elements in matrix!" << std::endl;
		}
		else if (numThreads_block > 1024) {
		     std::cout << "Error -- there are too many threads in block!" << std::endl;
		}

	} while (numThreads_x < N || numThreads_block > 1024);

	dim3 Grid(Grid_Dim_x, Grid_Dim_y);	//Grid structure
	dim3 Block(Block_Dim_x,Block_Dim_y);	//Number of threads per block.

/*
This section will dynamically allocate the memory needed
for the matrices in both the cpu and gpu calculations.
Here we will also be using fillMatrices from matdefine in
order to populate our two matrices.
*/

	hipMallocManaged( (void**) &a, N * N * sizeof(int));
	hipMallocManaged( (void**) &b, N * N * sizeof(int));
	hipMallocManaged( (void**) &c, N * N * sizeof(int));
//	hipMallocManaged( (void**) &d, N * N * sizeof(int));
	d = (int*) malloc(N * N * sizeof(int));
	
	fillMatrices(a,b,N);			// used to generate the arrays, found in matdefine.cu
	
	std::cout << "Array A" << std::endl;
	printMatrix(a, N);			// used to display matrix A, used in order to verify what was in the matrix for debugging
	std::cout << "Array B" << std::endl;
	printMatrix(b, N);			// used to display matrix B, used in order to verify what was in the matrix for debugging

/*
In this section we will be performing the nececcary steps in
order to run our computaion on the GPU. The hipEventCreate is
used to created the events for our timers. hipMemcpy is used
to copy the matrix from the host to the device, which is then
used in the matgpuadd function, which used the entered results
from before, to specify the number of blocks and the number of
threads per block that will be used on the GPU
*/

	matgpumult<<<Grid,Block>>>(a,b,c,N);
        std::cout << "Array C" << std::endl;
	printMatrix(c, N);

/*
In this section we will be perofming the necessary steps
to run the sequential computations on the CPU
*/

	matcpumult(a,b,d,N);		// do calculation on the cpu

        std::cout << std::endl; 
	std::cout << "Checking if the results from the cpu calculation = gpu  calculation" << std::endl;

	for(int i = 0;i < N*N;i++) {  // checking if the matrix from the gpu is the same as cpu
		if (c[i] != d[i] ) { 
			std::cout << "ERROR results are not equal" << std::endl;
			break;
		}
	}
	
/*
Performing methods to free allocated memory
*/
	hipFree(a);
	hipFree(b);
	hipFree(c);
	free(d);

	std::cout << "To continue type c, to end press ctrl-z" << std::endl;
	std::cin >> check;
} while(check == 'c');
	return 0;
}
