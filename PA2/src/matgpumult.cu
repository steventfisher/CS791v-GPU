#include "hip/hip_runtime.h"

#include "matgpumult.h"

__global__ void matgpumult(int *a, int *b, int *c, int N) {
    int k;
    int sum = 0;
    
    for(k = 0; k < N; ++k) {
	sum +=  a[threadIdx.y * N + k] + b[k * N + threadIdx.x];
    }
    c[threadIdx.y*N + threadIdx.x] = sum;
}