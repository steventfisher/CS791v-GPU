#include "hip/hip_runtime.h"

#include "matgpumult.h"

__global__ void matgpumult(int *a, int *b, int *re, int N) {
    int k;
    int sum = 0;
    
    for(k = 0; k < N; k++) {
	sum +=  a[threadIdx.y * N + k] * b[k * N + threadIdx.x];
    }
    re[threadIdx.y * N + threadIdx.x] = sum;
}