#include "hip/hip_runtime.h"
#include "knngpu.h"
#include <math.h>
#include <algorithm>

__global__ void knngpu(float *a, float *b, int N)
{
	
  int i, j, k, l, m;
  float tmpdst = 0;
  //int q = 3;
  //int count = 0;
  float tmp = 0;
  float c[N];

  for(i = threadIdx.x + blockDim.x*blockIdx.x; i < N; i += blockDim.x*gridDim.x){
      for(j = threadIdx.y + blockDim.x * blockIdx.y; j < N; j += blockDim.y*gridDim.y){
          if(a[i*N + j] == -99999){
	      for(k = 0; k < N; ++k){
                  tmp = 0;
                  if(k != i){
	              for(l = 2; l < N; ++l){
                          tmp += pow(a[k*N + l]-a[i*N + l],2);
		      }
                  }
                  b[k] = sqrt(tmp);
              }
	      for(k = 0; k < N; ++k){
                  for(l = 0; l < N; ++l){
                      if(b[j] > b[k]){
                      }
                  }
              }	      
	  }
      }
      
  }
}
