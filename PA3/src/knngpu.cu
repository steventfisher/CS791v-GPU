#include "hip/hip_runtime.h"

#include "knngpu.h"
#include <math.h>
#include <algorithm>

    __global__ void knngpu(float *a, float *b, int N)
{
	
  int i, j, k, l, m;
  float tmpdst = 0;
  //int q = 3;
  //int count = 0;
  float tmp = 0;

  for(i = threadIdx.x + blockDim.x*blockIdx.x; i < N; i += blockDim.x*gridDim.x){
      for(j = threadIdx.y + blockDim.x * blockIdx.y; j < N; j += blockDim.y*gridDim.y){
          if(a[i*N + j] == -99999){
	      for(k = 0; k < N; ++k){
	          for(j = 0; j < N; ++j){
                  b[k] += 1;
		  }
              }	      
	  }
      }
      
  }
}
