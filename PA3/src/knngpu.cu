#include "hip/hip_runtime.h"

#include "knngpu.h"
#include <math.h>

    __global__ void knnGpu(float *A, float *B, int Size)
{
    //int x = threadIdx.x + blockIdx.x * blockDim.x;
    //int y = threadIdx.y + blockIdx.y*blockDim.y;
    __syncthreads();
    
    float tmpdist = 0;
    for (int i = blockIdx.x*blockDim.x + threadIdx.x; i < Size*Size; i+= blockDim.x*gridDim.x){
       tmpdist = 0.0;
       if (A[i] == -99999) {
           for (int k = 0; k < Size; ++k) {
               if(k != i){
                 for (int j = 0; j < Size; ++j) {
                     tmpdist += 2;
                 }
               }
               __syncthreads();
               //B[k] = sqrt((float) tmpdist);
               B[k] = tmpdist;
            }            
       }
    }
    
  
}