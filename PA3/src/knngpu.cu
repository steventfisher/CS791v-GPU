#include "hip/hip_runtime.h"
#include "knngpu.h"
#include <math.h>
#include <algorithm>

__global__ void knngpu(float *a, float *b, int N)
{
	
  int i, j, k, l;
  float tmpdst = 0;
  int q = 5;
  int count = 0;
  float tmp = 0;

  for(i = threadIdx.x + blockDim.x*blockIdx.x; i < N; i += blockDim.x*gridDim.x){
      for(j = threadIdx.y + blockDim.x * blockIdx.y; j < N; j += blockDim.y*gridDim.y){
          if(a[i*N + j] == -99999){
	      for(k = 0; k < N; ++k){
                  tmp = 0;
                  if(k != i){
	              for(l = 2; l < N; ++l){
                          tmp += pow(a[k*N + l]-a[i*N + l],2);
		      }
                  }
                  b[k] = sqrt(tmp);
              }
	      for(k = 0; k < N; ++k){
                  for(l = 0; l < N; ++l){
                      if(b[l] > b[k]){
                          tmpdst = b[k];
                          b[k] = b[l];
                          b[l] = tmpdst;
                      }
                  }
              }
              tmpdst = 0;
              for(k = 0; k < N; ++k){
                  if(b[k] != 0 && count < q){
                      tmpdst += b[k];
		      count += 1;
                  } 
              }	      
              a[i*N + j] = tmpdst/q;

	  }
      }
      
  }
}
