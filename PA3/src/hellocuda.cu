/*
  In this program we will be using the GPU to add two square matrices. 

  This is the main program. You should also look at the header matgpuadd.h
  matcpuadd.h, and matdefine.h for the important declarations, and then look
  at matgpuadd.cu, matcpuadd.cu, and matdefine.cu to see how the methods were
  defined.

  AUTHOR: Steven Fisher
  CLASS: CS 791-GPU Computing
  ASSIGNMENT: PA3
 */

#include <iostream>

#include "knngpu.h"
#include "knncpu.h"
#include "knndefine.h"


int main() {
/*
This section is for the declaration of the variables that will be used
in our implementation of the matrix addition.
*/

	char check;

	int Grid_Dim_x=1, Grid_Dim_y=1;		//Grid structure values
	int Block_Dim_x=1, Block_Dim_y=1;	//Block structure values

	int numThreads_x;  			// number of threads available in device, each dimension
	int numThreads_block;			// number of threads in a block

	int N = 8;  				// size of array in each dimension
	char *a;
	//int *b,*c,*d;
/*
This section specifies the size limitations and allows the user to
specify the size of the matrices, the number of blocks used and the
number of threads per block to use.
*/
do{
	std::cout << "Maximum number of threads per block = 1024" << std::endl;
	std::cout << "Maximum sizes of the x and y dimensions of the thread block = 1024" << std::endl;
	std::cout << "Maximum size of each dimension of grid of thread blocks = 65535" << std::endl;

	do {
	   std::cout << "Enter the value(size of matrix) for N (5-129): ";
	   std::cin >> N;

	   if (N < 5) {
	      std::cout << "Error -- N has to be greater than 5!" << std::endl;
	   }
	   else if (N > 129) {
 	      std::cout << "Error -- N has to be less than or equal to 129!" << std::endl;
	   }
	} while ( N < 5 || N > 129);
	
	do {//Using a do while loop, since we want it to run at least once.
		std::cout << "Enter number of blocks per grid that will be used in both the x and y dimensions: ";
		std::cin >> Grid_Dim_x;

		Grid_Dim_y = Grid_Dim_x;  // square grid

		std::cout << "Enter number of threads that will used per block in both the x and y dimensions, currently " << Block_Dim_x << " (Needs to be < 32): ";
		std::cin >> Block_Dim_x;

		Block_Dim_y = Block_Dim_x;	//square blocks

		numThreads_x = Grid_Dim_x * Block_Dim_x;		// total number of threads in x dimension
		
		numThreads_block = Block_Dim_x * Block_Dim_y;	// number of threads in a block

		if (numThreads_x < N) {
		   std::cout <<"Error -- number of threads in the x or y dimensions is less than thenumber of elements in matrix!" << std::endl;
		}
		else if (numThreads_block > 1024) {
		     std::cout << "Error -- there are too many threads in block!" << std::endl;
		}

	} while (numThreads_x < N || numThreads_block > 1024);
	
	a = (char *) malloc(N*N);
	readCsv(*a, N);

	dim3 Grid(Grid_Dim_x, Grid_Dim_y);	//Grid structure
	dim3 Block(Block_Dim_x,Block_Dim_y);	//Number of threads per block.

/*
This section will dynamically allocate the memory needed
for the matrices in both the cpu and gpu calculations.
Here we will also be using fillMatrices from matdefine in
order to populate our two matrices.
*/

/*	hipMallocManaged( (void**) &a, N * N * sizeof(int));
	hipMallocManaged( (void**) &b, N * N * sizeof(int));
	hipMallocManaged( (void**) &c, N * N * sizeof(int));
	//d = (int*) malloc(N * N * sizeof(int));
	
	fillMatrices(c,b,N);			// used to generate the arrays
	
	std::cout << "Array A" << std::endl;
	printMatrix(a, N);			// used to display matrix A, used in order to verify what was in the matrix for debugging
	std::cout << "Array B" << std::endl;
	printMatrix(b, N);			// used to display matrix B, used in order to verify what was in the matrix for debugging
*/
/*
In this section we will be performing the nececcary steps in
order to run our computaion on the GPU. The hipEventCreate is
used to created the events for our timers. hipMemcpy is used
to copy the matrix from the host to the device, which is then
used in the matgpuadd function, which used the entered results
from before, to specify the number of blocks and the number of
threads per block that will be used on the GPU
*/

/*	matgpumult<<<Grid,Block>>>(a,b,c,N);
	hipDeviceSynchronize();
        std::cout << "Array C" << std::endl;
	printMatrix(c, N);
/*
/*
In this section we will be perofming the necessary steps
to run the sequential computations on the CPU
*/

/*	matcpumult(a,b,d,N);		// do calculation on the cpu
        std::cout << "Array D" << std::endl;
	printMatrix(d, N);	

        std::cout << std::endl; 
	std::cout << "Checking if the results from the cpu calculation = gpu  calculation" << std::endl;

	for(int i = 0;i < N*N;i++) {  // checking if the matrix from the gpu is the same as cpu
		if (c[i] != d[i] ) { 
			std::cout << "ERROR results are not equal" << std::endl;
			break;
		}
	}
*/	
/*
Performing methods to free allocated memory
*/
/*	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);
*/
	free(a);
	std::cout << "To continue type c, to end press q" << std::endl;
	std::cin >> check;
} while(check == 'c');
	return 0;
}
