#include "hip/hip_runtime.h"

#include "knndefine.h"
#include <iostream>
#include <math.h>

void fillMatrices(int *a, int *b, int N) {

	int rows, columns;
	int count = 0;
	int countEnd = floor(N * 0.1);

	srand(1);
	
	for(rows=0; rows < N; rows++)		// load arrays with some numbers
	   for(columns=0; columns < N; columns++) {
		a[rows * N + columns] = rand() %10;
		b[rows * N + columns] = 0;
	}

	while (count < countEnd){
            randChange = rand() % N;
	    if(a[randChange * N + 2] != -99999){
	        a[rows * N + columns] = -99999;
		count += 1;
	    }
	    
	}
}

void readCsv(int *a, int numEntry) {
std::ifstream fin;
    std::string temp;

    std::cout << std::endl << std::endl << "Reading in CSV Data..." << std::endl;

    fin.open("../data/PA3_nrdc_datav2.csv");

    // read all the AS data from the file
    for(int i = 0; i < numEntry; i++)
    {
        fin >> a[i];
    }

    // close the file
    fin.close();

    std::cout << "Finished Reading in CSV Data" << std::endl << std::endl;
}

void printMatrix(int *h, int N) {

	std::cout << "Array, First 2 columns" << std::endl;

	for (int row = 0; row < 2; row += 1) {
	  for (int col = 0; col < N; col += 1) 
	 	std::cout << h[col + row * N] << "\t";
	  std::cout << std::endl;
	}
}