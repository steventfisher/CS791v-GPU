#include "hip/hip_runtime.h"

#include "knndefine.h"
#include <iostream>
#include <math.h>
#include <time.h>

void fillMatrices(float *a, float *b, int N) {

	int rows, columns;
	int count = 0;
	int countEnd = floor(N * 0.1);

	srand(time(NULL));
	
	for(rows=0; rows < N; rows++)		// load arrays with some numbers
	   for(columns=0; columns < N; columns++) {
		a[rows * N + columns] = rand() %10;
	}

	for(rows = 0; rows < N; rows++){
	    b[rows] = 0;
	}
	if(countEnd == 0){
	    countEnd = 1;
	}
	std::cout << "CountEnd: " << countEnd << std::endl;
	while (count < countEnd){
	    srand(time(NULL));
            int randChange = rand() % N;
	    if(a[randChange * N + 1] != -99999){
	        std::cout << "randChange: " << randChange << std::endl;
	        a[randChange * N + 1] = -99999;
		count += 1;
	    }
	    
	}
}


/*void readCsv(float *a, int numEntry) {
    std::ifstream fin;
    std::string temp;

    std::cout << std::endl << std::endl << "Reading in CSV Data..." << std::endl;

    fin.open("../data/PA3_nrdc_datav2.csv");

    // read all the AS data from the file
    for(int i = 0; i < numEntry; i++)
    {
        fin >> a[i];
    }

    // close the file
    fin.close();

    std::cout << "Finished Reading in CSV Data" << std::endl << std::endl;
}*/

void printMatrix(float *h, int N) {

	std::cout << "Array, First 2 columns" << std::endl;

	for (int row = 0; row < N; row += 1) {
	  for (int col = 0; col < N; col += 1) 
	 	std::cout << h[col + row * N] << "\t";
	  std::cout << std::endl;
	}
}

void printVector(float *h, int N){
	for (int row = 0; row < N; row++) {
	 	std::cout << h[row] << "\t";
	}
	std::cout << std::endl;

}