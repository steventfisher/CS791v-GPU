#include "hip/hip_runtime.h"

#include "knndefine.h"
#include <iostream>
#include <math.h>
#include <time.h>
#include <fstream>
#include <sstream>

void fillMatrices(float *b, int N) {

	int rows;

	for(rows = 0; rows < N; rows++){
	    b[rows] = 0;
	}
}

void randNan(float *a, int N){
    int count = 0;
    int countEnd = floor(N * 0.1);

    srand(time(NULL));

    if(countEnd == 0){
        countEnd = 1;
    }
    while (count < countEnd){
        srand(time(NULL));
        int randChange = rand() % N;
	if(a[randChange * N + 1] != -99999){
	    a[randChange * N + 1] = -99999;
	    count += 1;
	}
	    
    }
    
}

void copyMatrix(float *a, float *b, int N){
    for (int i = 0; i < N; ++i){
        for (int j = 0; j < N; ++j){
            b[i*N + j] = a[i*N + j];
        }
    }
}

void readCsv(float *a, int numEntry) {
    std::ifstream fin;
    std::string temp;

    std::cout << std::endl << std::endl << "Reading in CSV Data..." << std::endl;

    fin.open("../src/PA3_nrdc_datav2.csv");

    // read all the AS data from the file
    for(int row = 0; row < numEntry; ++row){
        std::string line;
        std::getline(fin, line);
	if (!fin.good())
	    break;

        std::stringstream iss(line);
	for(int col = 0; col < numEntry; ++col){
	    std::string val;
	    std::getline(iss, val, ',');
	    if(!iss.good())
	        break;
            std::stringstream convertor(val);
	    convertor >> a[row*numEntry + col];
	    if (col == 0){
	        a[row*numEntry + col] = row;
	    }
	}
    }
    

    // close the file
    fin.close();

    std::cout << "Finished Reading in CSV Data" << std::endl << std::endl;
}

void printMatrix(float *h, int N) {

	std::cout << "Array, First 2 columns" << std::endl;

	for (int row = 0; row < N; row += 1) {
	  for (int col = 0; col < N; col += 1) 
	 	std::cout << h[col + row * N] << "\t";
	  std::cout << std::endl;
	}
}

void printVector(float *h, int N){
	for (int row = 0; row < N; row++) {
	 	std::cout << h[row] << "\t";
	}
	std::cout << std::endl;

}
