#include "hip/hip_runtime.h"

#include "knncpu.h"
#include <math.h>
#include <iostream>
#include <algorithm>

void knncpu(float *a, float *b, int N) {

  int i, j, k, l, m;
  float tmpdst = 0;
  int q = 5;
  int count = 0;
  float tmp = 0;

  for(i = 0; i < N; ++i){
      for(j = 0; j < N; ++j){
          if(a[i*N + j] == -99999){
	      std::cout << "Found" << std::endl;
	      for(k = 0; k < N; ++k){
	          tmp = 0;
	          for(l = 2; l < N; ++l){
		      tmp += pow(a[k*N + l] - a[i*N + l],2);
		  }
		  b[k] = sqrt(tmp);
	      }
//	      std::cout << "tmp: " << tmp << std::endl;
	      std::sort(b, b + N);
	      tmpdst = 0;
	      for(m = 0; m < N; ++m){
	          if(b[m] != 0 && count < q){
	              tmpdst += b[m];
		      count += 1;
	          }
	      }
	      a[i*N + j] = tmpdst/q;
	  }
      }
  }

}
